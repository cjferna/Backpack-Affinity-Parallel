/*
  CPP_CONTEST=2013Cal-2011
  CPP_PROBLEM=S
  CPP_LANG=CUDA
  CPP_PROCESSES_PER_NODE=saturno 1
*/

/* RECORD
Carlos Javier Fernández Candel
student of Methodology of Parallel Programming
at the University of Murcia
February 1, 2016
time 8705 msec
speedup 1.33
The improvement is residual,
but it is included in the table records
because it is the first CUDA implementation*/

/*
 Tasks list algorithm scheme is used.
 Nodes are generated and saved on a list in parallel until level specified('LEVELS_TO_GENERATE'). That node list is 
 computed in parallel, where every thread takes as nodes as ('NUM_RESERVED_NODES'), to prevent critical waste time. 
 Finally, all threads tries to put his personal best result 'VOA' as global better result.
*/


#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <string.h>

#define LEVELS_TO_GENERATE 4 	// Numbers of levels to generate at start to balance compute distribution.
#define NODES_PER_THREAD 15		// Numbers of nodes per CUDA thread.

// Represent a node of solution.
struct Node {
	int *solution;			// solution[x]. Every posistion save the backpack where obejct 'x' is saved.
	int *backpacksWeights;	// backpacksWeights[x]. Weight of every backpack. Weight of backpack 'x'.
	int level;				// Actual level. Actual ojbect.
};

// Envolve a node. Put the actual object (level) on the next backpack.
__device__ void generate(struct Node *node, int *objectsWeights) {
	// If a object where in this position, his weight is subtracted.
	if (node->solution[ node->level ] != -1) {
		node->backpacksWeights[ node->solution[ node->level ] ] -= objectsWeights[ node->level ];
	}
	
	// Put the actual object (level) on the next backpack. His weight is added to the backpack.
	node->solution[ node->level ]++;
	node->backpacksWeights[ node->solution[ node->level ] ] += objectsWeights[ node->level ];
}

// Determine if node is a complete solution.
__device__ void isSolution(struct Node *node, int numbersOfObjects, int *backpacksCapacities, int *result) {
	// If it's last level. level == last level == last object.
	// And if the weight of the backpack where te actual object is saved (last level) is not over the backpack capacity. (Backpack where object is saved :solution[ level ] ]) 
	*result = ((node->level == numbersOfObjects - 1) 
		   && node->backpacksWeights[ node->solution[ node->level ] ] <= backpacksCapacities[ node->solution[ node->level ] ]);
}

// Value of this solution is calculated.
__device__ void calculateValue(int numbersOfObjects, struct Node *node, int *affinity, int *result) {
	int i, j;
	int value = 0;	// Result, value of this solution. Benefit.

	// For every object compared with the rest of objects.
	for (i = 0; i < numbersOfObjects; i++) {
		for (j = i + 1; j < numbersOfObjects; j++) {
			// If both object are in the same packack. solution[x] indicates backpack where object 'x' is saved.
			if (node->solution[i] == node->solution[j]) {
				// Value is added as affinity of both objects.
				value += (affinity[i*numbersOfObjects + j] + affinity[j*numbersOfObjects + i]);
			}
		}
	}
	
	*result = value;
}

// Determine  if a node can grown to the next level. 
__device__ void criteria(struct Node *node, int numbersOfObjects, int *backpacksCapacities, int *result) {
	// If the levels is not the last level.
	// And if the weight of the backpack where last object where saved is not over the capacity of the backpack.
	*result = (node->level != numbersOfObjects - 1 && node->backpacksWeights[ node->solution[ node->level ] ] <= backpacksCapacities[ node->solution[ node->level ] ]);
}

// Determine if this level has more brothers. 
__device__ void hasMoreBrothers(struct Node *node, int numbersOfBackpacks, int *result) {
	// If all backpacks are not used to try to save the current object (level).
	*result = (node->solution[ node->level ] < numbersOfBackpacks - 1);
}

// Deletes the actual object. Node level will be reduced.
__device__ void goBack(struct Node *node, int numbersOfBackpacks, int *objectsWeights) {
	// The weight of the last object(level) is subtracted.
	node->backpacksWeights[ numbersOfBackpacks - 1 ] -= objectsWeights[ node->level ];
	// The actual object is marked as not saved in a backpack.
	node->solution[ node->level ] = -1;
	// Level is reduced.
	node->level--;
}

// Base number to an exponent. To not link math.h -lm.
int raiseTo(int base, int exponente) {
	int result = 1;	
	
	int i;
	// Base is multiplied by base, 'exponente' times.
	for (i = 0; i < exponente; i++)
		result *= base;
	
	return result;
}

// Inicializate a solution. No one object is saved to a backpack. All backpacks weigth is zero.
void initializeSolutionSpace(int numbersOfBackpacks, int numbersOfObjects, struct Node *node) {	
	int i;	
	
	// No one object is saved to a backpack.
	for (i = 0; i < numbersOfObjects; i++)
		node->solution[i] = -1;

	// All backpacks weigth is zero.
	for (i = 0; i < numbersOfBackpacks; i++)
		node->backpacksWeights[i] = 0; 
	
	// The start level is zero.
	node->level = 0;
}

// Memory is allocated to save an solution(node). Objets assing array and Backpacks weights array.
void createSolutionSpace(int numbersOfBackpacks, int numbersOfObjects, struct Node **node) {
	// Struct memory allocated.
	*node = (struct Node*) malloc(sizeof(struct Node));
	
	// Starting level is zero.
	(*node)->level = 0;

	// All backpacks weigth is zero.
	(*node)->backpacksWeights = (int*) calloc (numbersOfBackpacks, sizeof(int));
	
	// No one object is saved to a backpack.
	(*node)->solution = (int*) malloc (sizeof(int) * numbersOfObjects);
	int i;
	for (i = 0; i < numbersOfObjects; i++)
		(*node)->solution[i] = -1;
}

// Free a node struct allocted memory. 
void destroySolutionSpace(struct Node *node) {		
	free(node->solution);
	free(node->backpacksWeights);
	free(node);
}

// Alocate memory for tasks array.
struct Node** createTasksSpace(int numbersOfBackpacks) {
	// The array size is: backpacks ^ levels to generate. 
	int tamanoMaximoLista = raiseTo(numbersOfBackpacks, LEVELS_TO_GENERATE);
	// Array pointer.
	return (struct Node**) malloc(sizeof(struct Node*) * tamanoMaximoLista);		
}

// Free tasks array allocted memory. 
void freeTasksSpace(int generatedTasks, struct Node **tasksList) {
	int i;
	
	// All generated nodes are freed.
	for (i = 0; i < generatedTasks; i++) {
		destroySolutionSpace(tasksList[i]);	
	}
	
	free (tasksList);	// Tasks array is freed.
}

// Copy a node 'nodoOrigen' to 'nodoDestino'.
void nodeCopy(struct Node *nodoOrigen, struct Node *nodoDestino, int numbersOfObjects, int numbersOfBackpacks) {
	// Copy of actual solution.
	memcpy(nodoDestino->solution, nodoOrigen->solution, sizeof(int) * numbersOfObjects);
	// Copy of backpacks weights.
	memcpy(nodoDestino->backpacksWeights, nodoOrigen->backpacksWeights, sizeof(int) * numbersOfBackpacks);
	// Copy of level.
	nodoDestino->level = nodoOrigen->level;
}

// Inicializate a node saving the actual object (level) to the specified backpack (mochila).
void InitializeNode(struct Node * node, int level, int backpack, int *objectsWeights) {
	// Lelvel is specified.
	node->level = level;	
	
	// If a object where in this position, his weight is subtracted.
	if (node->solution[ node->level ] != -1) {
		node->backpacksWeights[ node->solution[ node->level ] ] -= objectsWeights[ node->level ];
	}

	// Put the actual object (level) on the specified backpack. 
	node->solution[ node->level ] = backpack;
	// His weight is added to the backpack.
	node->backpacksWeights[ node->solution[ node->level ] ] += objectsWeights[ node->level ];
	// Level is increased. The object (level) is saved.
	node->level++;
}

void generate_generate(struct Node *node, int *objectsWeights) {
	// If a object where in this position, his weight is subtracted.
	if (node->solution[ node->level ] != -1) {
		node->backpacksWeights[ node->solution[ node->level ] ] -= objectsWeights[ node->level ];
	}
	
	// Put the actual object (level) on the next backpack. His weight is added to the backpack.
	node->solution[ node->level ]++;
	node->backpacksWeights[ node->solution[ node->level ] ] += objectsWeights[ node->level ];
}

// Determine  if a node can grown to the next level. 
int generate_criteria(struct Node *node, int numbersOfObjects, int *backpacksCapacities) {
	// If the levels is not the last level.
	// And if the weight of the backpack where last object where saved is not over the capacity of the backpack.
	return (node->level != numbersOfObjects - 1 && node->backpacksWeights[ node->solution[ node->level ] ] <= backpacksCapacities[ node->solution[ node->level ] ]);
}

// Generate node of the specified level. If level is target level, the node is saved in tasks array. Otherwise, childrens are generated.
void createTasks(struct Node * raiz, struct Node** tasksList, int nivelActual, 
				 int nivelesGenerar, int numbersOfBackpacks, int numbersOfObjects, int *objectsWeights, int *backpacksCapacities, int *generatedTasks) {
	// Si no level must be generated, return.
	if (nivelesGenerar == 0) {
		return;
	}
	// If target level is over or equal to the object numbers, levels generated are object - 1.
	else if (nivelesGenerar >= numbersOfObjects) {
		nivelesGenerar = numbersOfObjects - 1;
	}
	
	// Generation tasks variables.
	struct Node *actualNode, *lastNode;								// ActualNode and LastNodeGenerated
	createSolutionSpace(numbersOfBackpacks, numbersOfObjects, &actualNode);	// Memory allocated.
	nodeCopy(raiz, actualNode, numbersOfObjects, numbersOfBackpacks);		// Copy of root node into actualNode.
	
	int m;	
	// For every backpack.
	for (m = 0; m < numbersOfBackpacks; m++) {
		// Node is generated. Object (level) is saved into next backpack.
		generate_generate(actualNode, objectsWeights); 
		
		// If node meets criteria, this node can be saved as task or ready to generate childrens.
		if (generate_criteria(actualNode, numbersOfObjects, backpacksCapacities)) {
			// ActualNode is now LastNode.
			lastNode = actualNode;
			createSolutionSpace(numbersOfBackpacks, numbersOfObjects, &actualNode);		// Memory is allocated.
			nodeCopy(lastNode, actualNode, numbersOfObjects, numbersOfBackpacks);   	// Node copy.
			
			// Level is increased.
			lastNode->level++;					
			
			// If level generated is not the target level. Childrens are generated by recursivity.
			if (nivelActual + 1 < nivelesGenerar) {
				createTasks(lastNode, tasksList, nivelActual+1, nivelesGenerar, numbersOfBackpacks, numbersOfObjects, objectsWeights, backpacksCapacities, generatedTasks);
			// Otherwise, this node is saved in task list.
			} else if (nivelActual + 1 == nivelesGenerar) {			
				// Node is saved as task. Array index is increased.
				tasksList[*generatedTasks] = lastNode;					
				(*generatedTasks)++;
			}
		}
	}
}

// Tasks are generated in parallel. 
void generateTasks(int numbersOfBackpacks, int numbersOfObjects, int *backpacksCapacities, int *objectsWeights, int *generatedTasks, struct Node **tasksList) {
	struct Node *node;
	createSolutionSpace(numbersOfBackpacks, numbersOfObjects, &node);		
	
	int i;
	for (i = 0; i < numbersOfBackpacks; i++) {
		// First level node is generated.
		InitializeNode(node, 0, i, objectsWeights);		
		// Childrens are generated.
		createTasks(node, tasksList, 1, LEVELS_TO_GENERATE, numbersOfBackpacks, numbersOfObjects, objectsWeights, backpacksCapacities, generatedTasks);
	}
	
	// Generated node is freed.
	destroySolutionSpace(node);	
}

// Sequential Backtracking for a node.
__device__ void backtracking(struct Node *node, int *bestPersonalValue, int numbersOfBackpacks, int numbersOfObjects, int *backpacksCapacities, int *objectsWeights, int *affinity) {
	// Initial level of the node.
	int initialLevel = node->level;
	
	do {
		generate(node, objectsWeights); 

		// If node is a solution. His value is calculated and it's compared with best value.
		int isSolutionVar;
		isSolution(node, numbersOfObjects, backpacksCapacities, &isSolutionVar);
		if (isSolutionVar) {
			int value;
			calculateValue(numbersOfObjects, node, affinity, &value);
			if (value > *bestPersonalValue) {
				*bestPersonalValue = value;
			}
		}
		
		// If this node meets the criteria, level is increased.
		int criteriaVar = 0;
		criteria(node, numbersOfObjects, backpacksCapacities, &criteriaVar);
		
		if (criteriaVar) {
			node->level++;
		}
		// Otherwhise, while level is diferent of initial task level and the node has no brothers (more backpacks to save the actual objet), level is decreased. 
		else {
			int more;
			hasMoreBrothers(node, numbersOfBackpacks, &more);
			
			while (node->level >= initialLevel && !more) {
				goBack(node, numbersOfBackpacks, objectsWeights);
				hasMoreBrothers(node, numbersOfBackpacks, &more);
			}
		}
	// While initial level is not reached.
	} while (node->level >= initialLevel);
}

// Compute all task of tasks array in parallel. 
__global__ void computeTasks(int numbersOfBackpacks, int numbersOfObjects, int *backpacksCapacities, int *objectsWeights, int *affinity,
					int generatedTasks, struct Node **tasksList, int *topValues) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;		

	if (idx  < generatedTasks) {
		tasksList[idx]->level = LEVELS_TO_GENERATE;
		topValues[idx] = 0;
		backtracking(tasksList[idx], &topValues[idx], numbersOfBackpacks, numbersOfObjects, backpacksCapacities, objectsWeights, affinity);
	}
}

void setCudaMemory(int **dev_backpacksCapacities, int **dev_objectsWeights, int **dev_affinity, int *backpacksCapacities, 
				   int *objectsWeights, int *affinity, int numbersOfBackpacks, int numbersOfObjects) {
	// Memory Set.	
	hipMalloc((void**)dev_backpacksCapacities, numbersOfBackpacks * sizeof(int));	
	hipMalloc((void**)dev_objectsWeights, numbersOfObjects * sizeof(int));	
	hipMalloc((void**)dev_affinity, numbersOfObjects * numbersOfObjects * sizeof(int));
	
	// Copies	
	hipMemcpy(*dev_backpacksCapacities, backpacksCapacities, numbersOfBackpacks * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(*dev_objectsWeights, objectsWeights, numbersOfObjects * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(*dev_affinity, affinity, numbersOfObjects * numbersOfObjects * sizeof(int), hipMemcpyHostToDevice);
}

void setAndCopyTaskListToCudaMemory(struct Node ***dev_tasksList, struct Node **h_tasksList, struct Node **tasksList, struct Node **h_tasksList_to_free, int generatedTasks,
									int numbersOfBackpacks, int numbersOfObjects) {
	hipMalloc((void**)dev_tasksList, sizeof(struct Node*) * generatedTasks);
	
	int i;
	for(i = 0; i < generatedTasks; i++) {
		// Struct Memory.
		struct Node *nodo = (struct Node*) malloc(sizeof(struct Node));	
		h_tasksList_to_free[i] = nodo;
		
		// BackPackWeights Memory.
		hipMalloc((void**) &nodo->backpacksWeights, numbersOfBackpacks * sizeof(int));
		hipMemcpy(nodo->backpacksWeights, tasksList[i]->backpacksWeights, numbersOfBackpacks * sizeof(int), hipMemcpyHostToDevice);
		
		// Solution Memory.
		hipMalloc((void**) &nodo->solution, numbersOfObjects * sizeof(int));
		hipMemcpy(nodo->solution, tasksList[i]->solution, numbersOfObjects * sizeof(int), hipMemcpyHostToDevice);
		
		
		hipMalloc((void**)&h_tasksList[i], sizeof(struct Node));
		hipMemcpy(h_tasksList[i], nodo, sizeof(struct Node), hipMemcpyHostToDevice);
	}
	
	hipMemcpy(*dev_tasksList, h_tasksList, sizeof(struct Node*) * generatedTasks, hipMemcpyHostToDevice);
}


// Compare si the personal best actual value is betther than the global best value. 
void setLocalBestValue(int *result, int *h_topValues, int generatedTasks) {
	int i;
	for(i = 0; i < generatedTasks; i++) {
		if ((*result) < h_topValues[i]) {
			(*result) = h_topValues[i];
		}
	}
}

void freeCudaMemory(int *dev_topValues, struct Node **dev_tasksList, int *dev_backpacksCapacities, int *dev_objectsWeights, int *dev_affinity, 
					struct Node **h_tasksList, struct Node **h_tasksList_to_free, int *h_topValues, int generatedTasks) {
	int i;
	for(i = 0; i < generatedTasks; i++) {
		hipFree(h_tasksList_to_free[i]->backpacksWeights);
		hipFree(h_tasksList_to_free[i]->solution);
		hipFree(h_tasksList[i]);
	}
	
	hipFree(dev_topValues);
	hipFree(dev_tasksList);
	hipFree(dev_backpacksCapacities);
	hipFree(dev_objectsWeights);
	hipFree(dev_affinity);
	
	free(h_tasksList_to_free);
	free(h_tasksList);
	free(h_topValues);
}

int sec(int numbersOfBackpacks, int numbersOfObjects, int *backpacksCapacities, int *objectsWeights, int *affinity) {
	
/** Generate Tasks*/ 
	struct Node **tasksList = createTasksSpace(numbersOfBackpacks);	// Tasks array.
	int generatedTasks = 0;
	generateTasks(numbersOfBackpacks, numbersOfObjects, backpacksCapacities, objectsWeights, &generatedTasks, tasksList);

/** CUDA Memory and Copies */
	int *dev_backpacksCapacities = 0, *dev_objectsWeights = 0, *dev_affinity = 0;	
	setCudaMemory(&dev_backpacksCapacities, &dev_objectsWeights, &dev_affinity, backpacksCapacities, objectsWeights, affinity, numbersOfBackpacks, numbersOfObjects);
	
	struct Node **dev_tasksList = 0, **h_tasksList = (struct Node**) malloc(sizeof(struct Node*) * generatedTasks);
	struct Node **h_tasksList_to_free = (struct Node**) malloc(sizeof(struct Node*) * generatedTasks);
	
	setAndCopyTaskListToCudaMemory(&dev_tasksList, h_tasksList, tasksList, h_tasksList_to_free, generatedTasks, numbersOfBackpacks, numbersOfObjects);	
	int *dev_topValues = 0, *h_topValues = (int *) malloc(generatedTasks * sizeof(int));	
	hipMalloc((void**)&dev_topValues, generatedTasks * sizeof(int));
	
/** Tasks compute */
	int numThreads = NODES_PER_THREAD, numBlocks = (generatedTasks / NODES_PER_THREAD) +1;
	computeTasks<<<numBlocks, numThreads, 0>>>(numbersOfBackpacks, numbersOfObjects, dev_backpacksCapacities, dev_objectsWeights, dev_affinity, generatedTasks,
											   dev_tasksList, dev_topValues);
/* Result Set */
	int result = 0;
	hipMemcpy(h_topValues, dev_topValues, generatedTasks * sizeof(int), hipMemcpyDeviceToHost);
	setLocalBestValue(&result, h_topValues, generatedTasks);

/* Memory Free */
	freeCudaMemory(dev_topValues, dev_tasksList, dev_backpacksCapacities, dev_objectsWeights, dev_affinity,h_tasksList, h_tasksList_to_free, h_topValues, generatedTasks);	
	freeTasksSpace(generatedTasks, tasksList);
	
	return result;
}
